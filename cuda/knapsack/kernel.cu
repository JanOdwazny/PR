#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <algorithm>
#include <random>
#include <time.h>
#include <hip/hip_runtime.h>
#include <chrono>

int CAPACITY[] = { 10000, 20000, 30000,40000, 50000,60000 };
int NUM_ITEMS[] = { 5000, 10000, 15000, 30000, 40000, 50000 };

using namespace std::chrono;




__global__ void knapsackKernel(int* d_weights, int* d_values, int* d_dp, int num_items, int capacity) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > capacity) return;

    for (int item = 1; item <= num_items; ++item) {
        if (i >= d_weights[item - 1]) {
            int included = d_values[item - 1] + d_dp[(item - 1) * (capacity + 1) + (i - d_weights[item - 1])];
            int excluded = d_dp[(item - 1) * (capacity + 1) + i];
            d_dp[item * (capacity + 1) + i] = fmax(included, excluded);
        }
        else {
            d_dp[item * (capacity + 1) + i] = d_dp[(item - 1) * (capacity + 1) + i];
        }
    }

}

int main() {
    srand(time(NULL));
    /*const int num_items = 10000;*/
    
    //int values[num_items] = { 0 };
    //int weights[num_items] = { 0 };
    int capacity = 10000;

    for (int i = 0; i < 6; i++) {
        //int capacity = CAPACITY[i];
        int num_items = NUM_ITEMS[i];
        int* values = new int[num_items];
        int* weights = new int[num_items];
        
        
        for (int j = 0; j < num_items; j++)
        {
            values[j] = rand() % 100 + 10;
            weights[j] = rand() % (capacity / 2) + 1;
        }
        

        int size = (num_items + 1) * (capacity + 1);
        std::vector<int> dp(size, 0);

        int* d_weights, * d_values, * d_dp;
        hipMalloc((void**)&d_weights, num_items * sizeof(int));
        hipMalloc((void**)&d_values, num_items * sizeof(int));
        hipMalloc((void**)&d_dp, size * sizeof(int));

        hipMemcpy(d_weights, weights, num_items * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_values, values, num_items * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dp, dp.data(), size * sizeof(int), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (capacity + blockSize - 1) / blockSize;

       
        auto start = high_resolution_clock::now();
        
        
        knapsackKernel << <numBlocks, blockSize >> > (d_weights, d_values, d_dp, num_items, capacity);
       
        hipDeviceSynchronize();

       
        auto stop = high_resolution_clock::now();
        auto duration = duration_cast<milliseconds>(stop - start);

        std::cout << capacity << ":" << num_items << ":" <<duration.count()<< std::endl;
        hipMemcpy(dp.data(), d_dp, size * sizeof(int), hipMemcpyDeviceToHost);

        

        hipFree(d_weights);
        hipFree(d_values);
        hipFree(d_dp);

        delete[] values;
        delete[] weights;
    }

    return 0;
}
